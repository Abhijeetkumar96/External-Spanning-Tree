#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#include "euler.cuh"
#include "cuda_utility.cuh"
#include "spanning_tree.cuh"

// #define DEBUG

__global__
void init(uint64_t* d_parentEdge, int* d_componentParent, int* d_rep, int nodes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < nodes) {
        d_parentEdge[idx] = INT_MAX;
        d_componentParent[idx] = idx;
        d_rep[idx] = idx;
    }
}

__global__ 
void HOOKING(
    long edges, 
    uint64_t* d_edgelist,
    int *rep, 
    int *componentParent, 
    bool isMaxIteration, 
    int *c_flag) {

    long tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(tid < edges) {
        
        uint64_t i = d_edgelist[tid];

        int u = i >> 32;  // Extract higher 32 bits
        int v = i & 0xFFFFFFFF; // Extract lower 32 bits
        
        int rep_u = rep[u];
        int rep_v = rep[v];

        if(rep_u != rep_v) {
            // 2 different components
            *c_flag = true;
            if(isMaxIteration) {
                componentParent[min(rep_u, rep_v)] = max(rep_u, rep_v);
            }
            else {
                componentParent[max(rep_u, rep_v)] = min(rep_u, rep_v);
            }
        }
    }
}

__global__ 
void UPDATE_REP_PARENT(int nodes, int *componentParent, int *rep) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < nodes) {
        if(rep[tid] == tid && componentParent[tid] != -1) {
            rep[tid] = componentParent[tid];
        }
    }
}

/**
 * Needs to be executed before UPDATE_REP_PARENT
 * @d_parentEdge : d_parentEdge[i] --> idx of the edge which connects ith tree to parent of ith tree
*/
__global__ 
void STORE_CROSS_EDGES(
    int edges,
    int *rep, 
    uint64_t* d_edgelist,
    int *componentParent, 
    uint64_t *d_parentEdge) {

    long tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < edges) {
       
        uint64_t i = d_edgelist[tid];

        int u = i >> 32;  // Extract higher 32 bits
        int v = i & 0xFFFFFFFF; // Extract lower 32 bits
        
        int rep_u = rep[u];
        int rep_v = rep[v];

        if(rep_u == rep_v)
            return;

        // printf("u = %d, v = %d , rep_u = %d, rep_v = %d \n", u, v, rep_u, rep_v);
        if( rep_v == componentParent[rep_u]){
            // u is the representative of the tree
            // v belongs to the parent tree of u

            d_parentEdge[rep_u] = d_edgelist[tid];
        }

        if( rep_u == componentParent[rep_v]) {
            d_parentEdge[rep_v] = d_edgelist[tid];
        }   
    }
}

__global__ 
void SHORTCUTTING(int nodes, int *rep, int *flag) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < nodes)
    {
        int prevValue = rep[tid];
        rep[tid] = rep[rep[tid]];
        if (prevValue != rep[tid])
        {
            *flag = 1;
        }
    }
}

void SpanningTree(
    int nodes, long edges,
    uint64_t* d_edgelist,
    int* d_rep,                     // rep[i] --> representative of the tree of which i is a part
    int* d_componentParent,         // componentParent[i] =rep of parent tree of the ith tree
    uint64_t* d_parentEdge,
    int* h_flag,                
    int* d_flag, 
    int* h_shortcutFlag,
    int* d_shortcutFlag) {

    #ifdef DEBUG
        std::cout << "Printing from hooking function:" << std::endl;

        std::cout << "nodes: " << nodes << ", edges: " << edges << std::endl;

        std::cout << "rep array:" << std::endl;
        // print_device_array(d_rep, nodes);

        std::cout << "Edges input to hooking: " << std::endl;
        print_device_edges(d_edgelist, edges);
    #endif

    int num_threads = 1024;

    int num_blocks_edges = (edges + num_threads - 1) / num_threads;
    int num_blocks_vert = (nodes + num_threads - 1) / num_threads;

    *h_flag = 1;
    *h_shortcutFlag = 1;
    int itr_count = 0;
    bool maxIteration = true;

        #ifdef DEBUG
            std::cout << "\n\n-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_\n\n";
            std::cout << "initial values:\n";
            // print rep and components parent
            std::cout << "Rep array:\n";
            print_device_array(d_rep, nodes);
            std::cout << "Components array:\n";
            print_device_array(d_componentParent, nodes);
            std::cout << "Selected Edges array:\n";
            print_device_edges(d_parentEdge, nodes);
            std::cout << "\n\n-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_\n\n";
        #endif

    while(*h_flag) {
        itr_count++;
        *h_flag = false;
        // hipMemcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind)
        CUDA_CHECK(hipMemcpy(d_flag, h_flag, sizeof(int), hipMemcpyHostToDevice), "Failed to copy flag to device");
        HOOKING<<<num_blocks_edges, num_threads>>> (
            edges,
            d_edgelist,
            d_rep,
            d_componentParent,
            maxIteration,
            d_flag);

        CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize after HOOKING");
        CUDA_CHECK(hipMemcpy(h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost), "Failed to copy flag back to device");

        maxIteration = !maxIteration;

        // !!! This should be done before updating
        STORE_CROSS_EDGES<<<num_blocks_edges, num_threads>>> (
            edges,
            d_rep,
            d_edgelist,
            d_componentParent,
            d_parentEdge
        );
        
        CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize after STORE_CROSS_EDGES");

        // rep[representative] = representative of its parent
        UPDATE_REP_PARENT<<<num_blocks_vert, num_threads>>> (
            nodes,
            d_componentParent,
            d_rep);
        CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize after UPDATE_REP_PARENT");
        
        *h_shortcutFlag = true;
        // auto start = std::chrono::high_resolution_clock::now();
        while(*h_shortcutFlag) {
            *h_shortcutFlag = false;
            CUDA_CHECK(hipMemcpy(d_shortcutFlag, h_shortcutFlag, sizeof(int), hipMemcpyHostToDevice), "Failed to copy h_shortcutFlag to device");
            SHORTCUTTING <<<num_blocks_vert, num_threads >>> (nodes, d_rep, d_shortcutFlag);
            CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize after SHORTCUTTING kernel");
            CUDA_CHECK(hipMemcpy(h_shortcutFlag, d_shortcutFlag, sizeof(int), hipMemcpyDeviceToHost), "Failed to copy back h_shortcutFlag to host");
        }

        #ifdef DEBUG
            int k = 0;
            std::cout << "\n\n-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_\n\n";
            std::cout << "Printing for " << k++ << " iteration.\n";
            // print rep and components parent
            std::cout << "Rep array:\n";
            print_device_array(d_rep, nodes);
            std::cout << "Components array:\n";
            print_device_array(d_componentParent, nodes);
            std::cout << "Selected Edges array:\n";
            print_device_edges(d_parentEdge, nodes);
            std::cout << "\n\n-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_\n\n";
        #endif
    }

    #ifdef DEBUG
        std::cout << "\n\n-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_\n\n";
        std::cout << "Printing Final Rep array:" << std::endl;
        print_device_array(d_rep, nodes);
        std::cout << std::endl;

        std::cout << "Printing spanning tree edges:" << std::endl;
        print_device_edges(d_parentEdge, nodes);
        std::cout << std::endl;
        std::cout << "\n\n-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_\n\n";
    #endif
}

void construct_spanning_tree(uint64_t* h_edgelist, int nodes, long edges) {

    long batch_size = 3;

    // The buffer to which we copy
    uint64_t* d_edgelist;
    CUDA_CHECK(hipMalloc(&d_edgelist, batch_size * sizeof(uint64_t)), "Allocation error");

    uint64_t* d_parentEdge;
    CUDA_CHECK(hipMalloc(&d_parentEdge, sizeof(uint64_t) * nodes), "Failed to allocate memory for d_parentEdge");
    
    int* d_componentParent;
    CUDA_CHECK(hipMalloc(&d_componentParent, sizeof(int) * nodes), "Failed to allocate memory for d_parentEdge");

    int* d_rep;
    CUDA_CHECK(hipMalloc(&d_rep, sizeof(int) * nodes), "Failed to allocate memory for d_rep_hook");

    int *h_flag;
    CUDA_CHECK(hipHostMalloc((void **)&h_flag, sizeof(int)), "Failed to allocate memory for c_flag");

    int *h_shortcutFlag;
    CUDA_CHECK(hipHostMalloc((void **)&h_shortcutFlag, sizeof(int)), "Failed to allocate memory for c_shortcutFlag");

    int *d_flag;
    CUDA_CHECK(hipMalloc((void **)&d_flag, sizeof(int)), "Failed to allocate memory for c_flag");

    int *d_shortcutFlag;
    CUDA_CHECK(hipMalloc((void **)&d_shortcutFlag, sizeof(int)), "Failed to allocate memory for c_shortcutFlag");

    int num_threads = 1024;
    int num_blocks_vert = (nodes + num_threads - 1) / num_threads;

    init<<<num_blocks_vert, num_threads>>>(d_parentEdge, d_componentParent, d_rep, nodes);
    CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize init kernel");

    auto start = std::chrono::high_resolution_clock::now();

    long num_batches = edges / batch_size;
    if (edges % batch_size != 0) {
        num_batches++;
    }

    for (long i = 0; i < num_batches; i++) {
        long start = i * batch_size;
        long end = std::min((i + 1) * batch_size, edges);
        long num_elements_in_batch = end - start;    

        CUDA_CHECK(hipMemcpy(
            d_edgelist, 
            h_edgelist + start, 
            num_elements_in_batch * sizeof(uint64_t), 
            hipMemcpyHostToDevice), 
        "Memcpy error");

        std::cout << "Batch " << i << ": " << start << " to " << end << std::endl;
        std::cout << "Number of elements in batch: " << num_elements_in_batch << std::endl;

        SpanningTree(
            nodes, num_elements_in_batch,
            d_edgelist, 
            d_rep, 
            d_componentParent, 
            d_parentEdge, 
            h_flag,
            d_flag,
            h_shortcutFlag,
            d_shortcutFlag);
    }

    CUDA_CHECK(hipMemcpy(h_flag, &d_rep[0], sizeof(int), hipMemcpyDeviceToHost), "Failed to copy flag from device to host");
    int root = *h_flag;
    std::cout << "Root Value: " << root << std::endl;

    cuda_euler_tour(nodes, root, d_parentEdge);

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration<double, std::milli>(stop - start).count();

    std::cout << "Spanning Tree construction took: " << duration << " ms.\n";

    CUDA_CHECK(hipFree(d_edgelist), "Failed to free d_edgelist");
}
